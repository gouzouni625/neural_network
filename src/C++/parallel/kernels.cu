#include "hip/hip_runtime.h"
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include "parameters.h"

__device__ __constant__ int deviceSizesOfLayers[NUMBER_OF_LAYERS];

// deviceSizesOfLayers has a constant value through the execution of the program.
// Load it on the constant memory of the GPU for faster reading.
void constantsInitialize(int *sizesOfLayers)
{
  if(hipMemcpyToSymbol(HIP_SYMBOL(deviceSizesOfLayers), sizesOfLayers, NUMBER_OF_LAYERS * sizeof(int)) != hipSuccess)
  {
    std::cout << "Error in hipMemcpyToSymbol for deviceSizesOfLayers..." << std::endl;
    exit(1);
  }
}

// Initializes nablaTheta to zero.
__global__
void initializeNablaThetaCU(int numberOfThetas, float *nablaTheta)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= 0 && index < numberOfThetas)
    nablaTheta[index] = 0;
}

// initializeNablaThetaCU wrapper.
void initializeNablaTheta(int numberOfThetas, float *deviceNablaTheta)
{
  // Each cuda thread will work on one variable.
  dim3 dimBlock(MAX_THREADS_PER_BLOCK, 1);
  dim3 dimGrid((int)(numberOfThetas / MAX_THREADS_PER_BLOCK) + 1, 1);
  initializeNablaThetaCU<<<dimGrid, dimBlock>>>(numberOfThetas, deviceNablaTheta);

  hipError_t err = hipDeviceSynchronize();  
  if(err != hipSuccess)  
  {
    std::cout << "Error in synchronizing for initializeNablaThetaCU " << err << std::endl;
    exit(1);
  }
}

// Neurons' activation function.
__device__
float sigmoid(float z)
{
  return (1 / (1 + exp(-z)));
}

// Calculates the output of each Neuron in the network given it's input.
__global__
void activations(float *trainingSet, int sample, float *weights, float *biases, int numberOfLayers,
                 int totalNeurons, int currentLayer, float *a)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x; // Which neuron am I working on?
  int currentLayerSize = deviceSizesOfLayers[currentLayer + 1];

  sample += blockIdx.y;

  int inputSize = deviceSizesOfLayers[currentLayer];
  int weightsIndex = index * inputSize;

  for(int i = 0;i < currentLayer;i++)
    index += deviceSizesOfLayers[i + 1];

  int aIndex = index + blockIdx.y * totalNeurons;

  int inputIndex = 0;
  float *input;
  if(currentLayer == 0)
  {
    input = trainingSet;
    inputIndex = sample * inputSize;
  }
  else
  {
    input = a;
    for(int i = 0;i < currentLayer - 1;i++)
      inputIndex += deviceSizesOfLayers[i + 1];
    inputIndex += blockIdx.y * totalNeurons;
  }

  // Each thread on this block is working on the same layer and thus using the same input.
  // Cooperatively load the input on the shared memory for faster reading.
  __shared__ float sharedInput[MAX_SIZE_OF_LAYER];
  float threadsAvailable = (MAX_THREADS_PER_BLOCK < currentLayerSize) ? MAX_THREADS_PER_BLOCK : currentLayerSize;
  for(int i = 0;i < (int)(inputSize / threadsAvailable) + 1;i++)
  {
    int tempIndex = threadIdx.x + i * threadsAvailable;
    if(tempIndex >= 0 && tempIndex < inputSize)
      sharedInput[tempIndex] = input[inputIndex + tempIndex];
  }

  __shared__ float sharedBiase;
  if(threadIdx.x == 0)
    sharedBiase = biases[index];

  for(int i = 0;i < currentLayer;i++)
    weightsIndex += deviceSizesOfLayers[i] * deviceSizesOfLayers[i + 1];

  __syncthreads();
  index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < 0 || index >= currentLayerSize)
    return;

  float sum = 0;
  for(int i = 0;i < inputSize;i++)
    sum += sharedInput[i] * weights[weightsIndex + i];

  a[aIndex] = sigmoid(sum + sharedBiase);
}

// Calculates the error of each neuron of the network.
__global__
void deltas(float *deltas, float *weights, float *a, int numberOfLayers, int currentLayer, int totalNeurons)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x; // Which neuron am I working on?
  int currentLayerSize = deviceSizesOfLayers[currentLayer + 1];

  int weightsIndex = index;

  for(int i = 0;i < currentLayer;i++)
    index += deviceSizesOfLayers[i + 1];

  int adIndex = index + blockIdx.y * totalNeurons;

  int deltasIndex = 0;

  for(int i = 0;i < currentLayer + 1;i++)
  {
    deltasIndex += deviceSizesOfLayers[i + 1];
    weightsIndex += deviceSizesOfLayers[i] * deviceSizesOfLayers[i + 1];
  }

  deltasIndex += blockIdx.y * totalNeurons;

  // Each thread on this block is working on the same layer and thus using the same delta values.
  // Cooperatively load these values to shared memory for faster reading.
  __shared__ float sharedDeltas[MAX_SIZE_OF_LAYER];
  int nextLayerSize = deviceSizesOfLayers[currentLayer + 2]; // How many deltas I want to load to shared memory.
  float threadsAvailable = (MAX_THREADS_PER_BLOCK < currentLayerSize) ? MAX_THREADS_PER_BLOCK : currentLayerSize;
  for(int i = 0;i < (int)(nextLayerSize / threadsAvailable) + 1;i++)
  {
    int tempIndex = threadIdx.x + i * threadsAvailable;
    if(tempIndex >= 0 && tempIndex < nextLayerSize)
      sharedDeltas[tempIndex] = deltas[deltasIndex + tempIndex];
  }

  __syncthreads();
  index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < 0 || index >= currentLayerSize)
    return;

  float sum = 0, aTemp = a[adIndex];
  for(int i = 0;i < nextLayerSize;i++)
   sum += weights[weightsIndex + i * currentLayerSize] * sharedDeltas[i];

  sum *= aTemp * (1 - aTemp);

  deltas[adIndex] = sum;
}

__global__
void firstDeltas(float *trainingLabels, int sample, float *a, float *deltas, int numberOfLayers, int totalNeurons)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x; // Which neuron am I working on?
  int currentLayerSize = deviceSizesOfLayers[numberOfLayers - 1];
  if(index < 0 || index >= currentLayerSize)
    return;

  sample += blockIdx.y; // Which sample am I working on? (There are several samples being fed concurrently to the
                        // network)

  int adIndex = index;
  for(int i = 0;i < numberOfLayers - 2;i++)
    adIndex += deviceSizesOfLayers[i + 1];

  adIndex += blockIdx.y * totalNeurons;

  deltas[adIndex] = a[adIndex] - trainingLabels[sample * currentLayerSize + index];
}

// Calculates the derivative of the const function with respect to each network's variable.
__global__
void nablas(float *trainingSet, int sample, int numberOfLayers, int totalNeurons, int concurrentSamples,
            float *nablaTheta, float *deltas, float *a)
{
  int neuron = blockIdx.x * blockDim.x + threadIdx.x; // Which neuron am I working on?

  int sum = 0, layer = numberOfLayers - 1;
  for(int i = 0;i < numberOfLayers - 2;i++)
  {
    sum += deviceSizesOfLayers[i + 1];
    if(sum > neuron)
    {
      layer = i + 1;
      break;
    }
  }

  int theta = blockIdx.y * blockDim.y + threadIdx.y; // Which variable am I working on?

  float *input;
  int inputIndex, inputCoefficient;
  if(layer == 1)
  {
    input = trainingSet;
    inputCoefficient = deviceSizesOfLayers[0];
    inputIndex = sample * deviceSizesOfLayers[0] + theta - 1;
  }
  else
  {
    input = a;
    inputCoefficient = totalNeurons;
    inputIndex = theta - 1;
    for(int i = 0;i < layer - 2;i++)
      inputIndex += deviceSizesOfLayers[i + 1];
  }

  int nablaThetaIndex = theta;
  int previousNeurons = 0;
  for(int i = 0;i < layer - 1;i++)
    previousNeurons += deviceSizesOfLayers[i + 1];
  for(int i = 0;i < neuron - previousNeurons;i++)
    nablaThetaIndex += deviceSizesOfLayers[layer - 1] + 1;
  for(int i = 0;i < layer - 1;i++)
    nablaThetaIndex += (deviceSizesOfLayers[i] + 1) * deviceSizesOfLayers[i + 1];

  // Every thread on this block is working on the same neuron and thus using the same deltas.
  // Cooperatively load these deltas on the shared memory for faster reading.
  __shared__ float sharedDeltas[MAX_CONCURRENT_SAMPLES];
  int inputSize = deviceSizesOfLayers[layer - 1], currentLayerSize = deviceSizesOfLayers[layer];
  float threadsAvailable = (MAX_THREADS_PER_BLOCK < currentLayerSize) ? MAX_THREADS_PER_BLOCK : currentLayerSize;
  for(int i = 0;i < (int)(concurrentSamples / threadsAvailable) + 1;i++)
  {
    int tempIndex = threadIdx.y + i * threadsAvailable;
    if(tempIndex >= 0 && tempIndex < concurrentSamples)
      sharedDeltas[tempIndex] = deltas[neuron + tempIndex * totalNeurons];
  }

  // Ensuring that all delta values have been loaded and dismissing the unneeded threads.
  __syncthreads();
  if(neuron < 0 || neuron >= totalNeurons)
    return;
  if(theta < 0 || theta > inputSize)
    return;

  // Calculating the derivate of the cost function with respect to the variable this thread is working on.
  float sum2 = nablaTheta[nablaThetaIndex];
  for(int i = 0;i < concurrentSamples;i++)
  {
    if(theta != 0)
      sum2 += input[inputIndex + i * inputCoefficient] * sharedDeltas[i];
    else
      sum2 += sharedDeltas[i];
  }

  nablaTheta[nablaThetaIndex] = sum2;
}

void backPropagation(float *deviceTrainingSet, float *deviceTrainingLabels, int numberOfSamples,
                     float* deviceWeights, float *deviceBiases, int numberOfLayers, int totalNeurons,
                     float *deviceNablaTheta, float *deviceActivations, float *deviceDeltas, int *sizesOfLayers,
                     int concurrentSamples)
{
  hipError_t err;
  for(int sample = 0;sample < numberOfSamples;sample += concurrentSamples)
  {
    // Calculating activations (feed forward).
    for(int i = 0;i < numberOfLayers - 1;i++) // For each layer...
    {
      // Each cuda thread works on a specific neuron and a specific sample.
      dim3 dimBlock(MAX_THREADS_PER_BLOCK, 1);
      dim3 dimGrid((int)(sizesOfLayers[i + 1] / MAX_THREADS_PER_BLOCK) + 1, concurrentSamples);
      activations<<<dimGrid, dimBlock>>>(deviceTrainingSet, sample, deviceWeights, deviceBiases, numberOfLayers,
                                         totalNeurons, i, deviceActivations);

      err = hipDeviceSynchronize();
      if(err != hipSuccess)
      {
        std::cout << "Error in synchronizing for activations " << err << std::endl;
        exit(1);
      }
    }

    // Calculate deltas (backpropagation).
    // Each cuda thread works on a specific neuron and a specific sample.
    dim3 dimBlock(MAX_THREADS_PER_BLOCK, 1);
    dim3 dimGrid((int)(sizesOfLayers[numberOfLayers - 1] / MAX_THREADS_PER_BLOCK) + 1, concurrentSamples);
    firstDeltas<<<dimGrid, dimBlock>>>(deviceTrainingLabels, sample, deviceActivations, deviceDeltas,
                                       numberOfLayers, totalNeurons);

    err = hipDeviceSynchronize();
    if(err != hipSuccess)
    {
      std::cout << "Error in synchronizing for firstDeltas " << err << std::endl;
      exit(1);
    }

    for(int i = numberOfLayers - 3;i >= 0;i--) // For each layer(beginning for the end, without the input layer).
    {
      dim3 dimBlock(MAX_THREADS_PER_BLOCK, 1);
      dim3 dimGrid((int)(sizesOfLayers[i + 1] / MAX_THREADS_PER_BLOCK) + 1, concurrentSamples);
      deltas<<<dimGrid, dimBlock>>>(deviceDeltas, deviceWeights, deviceActivations, numberOfLayers, i, totalNeurons);

      err = hipDeviceSynchronize();
      if(err != hipSuccess)
      {
        std::cout << "Error in synchronizing for deltas " << err << std::endl;
        exit(1);
      }
    }

      // Calculate nablaThetas.
      dim3 dimBlock2(1, MAX_THREADS_PER_BLOCK);
      int maxSizeOfLayer = sizesOfLayers[0];
      for(int i = 1;i < numberOfLayers - 1;i++)
        if(sizesOfLayers[i] > maxSizeOfLayer)
          maxSizeOfLayer = sizesOfLayers[i];

      // Each cuda thread works on a specific variable of a specific neuron.
      dim3 dimGrid2(totalNeurons, (int)((maxSizeOfLayer + 1) / MAX_THREADS_PER_BLOCK) + 1); // +1 for the biase.
      nablas<<<dimGrid2, dimBlock2>>>(deviceTrainingSet, sample, numberOfLayers, totalNeurons, concurrentSamples,
                                      deviceNablaTheta, deviceDeltas, deviceActivations);

    err = hipDeviceSynchronize();
    if(err != hipSuccess)
    {
      std::cout << "Error in synchronizing for nablas " << err << std::endl;
      exit(1);
    }
  }
}

// Updates network's variables using gradient descent.
__global__
void updateThetasCU(int trainingSetSize, int numberOfLayers, int totalNeurons, float *weights, float *biases,
                    float *nablaTheta, float gamma, float lambda)
{
  int neuron = blockIdx.x * blockDim.x + threadIdx.x; // Which neuron am I working on?
  if(neuron < 0 || neuron >= totalNeurons)
    return;

  int sum = 0, layer = numberOfLayers - 1;
  for(int i = 0;i < numberOfLayers - 2;i++)
  {
    sum += deviceSizesOfLayers[i + 1];
    if(sum > neuron)
    {
      layer = i + 1;
      break;
    }
  }

  int theta = blockIdx.y * blockDim.y; // Which variable am I working on?
  if(theta < 0 || theta > deviceSizesOfLayers[layer - 1])
    return;

  int nablaThetaIndex = theta;
  int previousNeurons = 0;
  for(int i = 0;i < layer - 1;i++)
    previousNeurons += deviceSizesOfLayers[i + 1];
  for(int i = 0;i < neuron - previousNeurons;i++)
    nablaThetaIndex += deviceSizesOfLayers[layer - 1] + 1;
  for(int i = 0;i < layer - 1;i++)
    nablaThetaIndex += (deviceSizesOfLayers[i] + 1) * deviceSizesOfLayers[i + 1];

  // Updating the variable.
  if(theta == 0) // I am a bias.
    biases[neuron] -= gamma * nablaTheta[nablaThetaIndex] / trainingSetSize;
  else
  {
    int weightsIndex = theta - 1;
    for(int i = 0;i < neuron - previousNeurons;i++)
      weightsIndex += deviceSizesOfLayers[layer - 1];
    for(int i = 0;i < layer - 1;i++)
      weightsIndex += deviceSizesOfLayers[i] * deviceSizesOfLayers[i + 1];
    weights[weightsIndex] -= gamma * ((nablaTheta[nablaThetaIndex] +
                                       lambda * weights[weightsIndex]) / trainingSetSize);
  }
}

// updateThetasCU wrapper.
void updateThetas(int trainingSetSize, int *sizesOfLayers, int numberOfLayers, int totalNeurons,
                  float *deviceWeights, float *deviceBiases, float *deviceNablaTheta, float gamma, float lambda)
{
  hipError_t err;
  dim3 dimBlock(MAX_THREADS_PER_BLOCK, 1);
  int maxSizeOfLayer = sizesOfLayers[0];
  for(int i = 1;i < numberOfLayers - 1;i++)
    if(sizesOfLayers[i] > maxSizeOfLayer)
      maxSizeOfLayer = sizesOfLayers[i];

  // Each cuda thread works on a specific variable of a specific neuron.
  dim3 dimGrid((int)(totalNeurons / MAX_THREADS_PER_BLOCK) + 1, maxSizeOfLayer + 1);
  updateThetasCU<<<dimGrid, dimBlock>>>(trainingSetSize, numberOfLayers, totalNeurons, deviceWeights,
                                        deviceBiases, deviceNablaTheta, gamma, lambda);

  err = hipDeviceSynchronize();
  if(err != hipSuccess)
  {
    std::cout << "Error in synchronizing updateThetasCU " << err << std::endl;
    exit(1);
  }
}

// Sets up cuda's random-number-generator.
__global__
void setupKernelCU(hiprandState *states, int processLoad)
{
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if(id < 0 || id >= processLoad)
    return;

  hiprand_init(id, id, 0, &states[id]);
}

// Distorts training set to virtually increase the size of it.
__global__
void distortCU(hiprandState *states, float *trainingSet, int processLoad)
{
  int image = blockIdx.x * blockDim.x + threadIdx.x; // Which image am I working on?
  if(image < 0 || image >= processLoad)
    return;

  float destortionType, parameter;
  destortionType = (hiprand_normal(&states[image]) + 1) / 2; // In what way should I destort the image?

  float trfMtx[2][2], pi = acosf(-1);
  if(destortionType < 0.4) // Rotating.
  {
    parameter = (hiprand_normal(&states[image]) / 12) * pi; // Angle.
    trfMtx[0][0] = cosf(parameter);
    trfMtx[0][1] = sinf(parameter);
    trfMtx[1][0] = -sinf(parameter);
    trfMtx[1][1] = cosf(parameter);
  }
  else if(destortionType < 0.7)  // Scaling.
  {
    parameter = (hiprand_normal(&states[image]) * 15 / 100) + 1; // Volume for horizontal axis.
    trfMtx[0][0] = 1 / parameter;
    trfMtx[0][1] = 0;
    trfMtx[1][0] = 0;
    parameter = (hiprand_normal(&states[image]) * 15 / 100) + 1; // Volume for vertical axis.
    trfMtx[1][1] = 1 / parameter;
  }
  else  // Shearing.
  {
    parameter = (hiprand_normal(&states[image]) * 15 / 100);
    trfMtx[0][0] = 1;
    trfMtx[0][1] = parameter;
    trfMtx[1][0] = 0;
    trfMtx[1][1] = 1;
  }

  image *= deviceSizesOfLayers[0];

  int size = deviceSizesOfLayers[0];
  float buffer[784];
  for(int i = 0;i < size;i++)
    buffer[i] = trainingSet[image + i];

  int index, l = sqrtf(size), xa, xb, yd, yb;
  float threshold = 1.0e-6, l1, l2, x, y, ca, cb, cc, cd;
  // Apply the randomly choosen transformation matrix to the image.
  // For each pixel on the transformed image, calculate the corresponding point on the initial image.
  // The pixel takes the color of the corresponding point which is calculated using linear interpolation.
  for(int xPrime = -l / 2;xPrime < l / 2;xPrime++)
  {
    for(int yPrime = -l / 2;yPrime < l / 2;yPrime++)
    {
      x = trfMtx[0][0] * xPrime + trfMtx[0][1] * yPrime;
      y = trfMtx[1][0] * xPrime + trfMtx[1][1] * yPrime;

      index = image + (l / 2 - 1 - yPrime) * l + (xPrime + l / 2);

      if(x < -l / 2 || x > l / 2 - 1 || y < -l / 2 || y > l / 2 - 1) // Point out of image bounds.
        trainingSet[index] = 0;
      else if((fabsf(x - l / 2 + 1) < threshold && fabsf(y - l / 2 + 1) < threshold))
        trainingSet[index] = buffer[l - 1];
      else if(fabsf(x - l / 2 + 1) < threshold)
      {
        x = l / 2 - 1;
        yd = floorf(y), yb = yd + 1;
        cb = buffer[(l / 2 - 1 - yb) * l + (int)(x + l / 2)];
        cd = buffer[(l / 2 - 1 - yd) * l + (int)(x + l / 2)];

        l1 = yb - y;
        trainingSet[index] = l1 * cd + (1 - l1) * cb;
      }
      else if(fabsf(y - l / 2 + 1) < threshold)
      {
        y = l / 2 - 1;
        xa = floorf(x), xb = xa + 1;
        ca = buffer[xa + l / 2];
        cb = buffer[xb + l / 2];

        l2 = xb - x;
        trainingSet[index] = l2 * ca + (1 - l2) * cb;
      }
      else
      {
        xa = floorf(x), xb = xa + 1;
        yd = floorf(y), yb = yd + 1;

        ca = buffer[(l / 2 - 1 - yb) * l + (xa + l / 2)];
        cb = buffer[(l / 2 - 1 - yb) * l + (xb + l / 2)];
        cc = buffer[(l / 2 - 1 - yd) * l + (xa + l / 2)];
        cd = buffer[(l / 2 - 1 - yd) * l + (xb + l / 2)];

        l1 = yb - y;
        l2 = xb - x;
        trainingSet[index] = l2 * (l1 * cc + (1 - l1) * ca) + (1 - l2) * (l1 * cd + (1 - l1) * cb);
      }
    }
  }
}

// distortCU wrapper.
void distort(hiprandState *deviceStates, float *deviceTrainingSet, int processLoad, bool setupKernel)
{
  hipError_t err;

  // Each cuda thread works on a specific image.
  dim3 dimBlock(MAX_THREADS_PER_BLOCK, 1);
  dim3 dimGrid((int)(processLoad / MAX_THREADS_PER_BLOCK) + 1, 1);

  if(setupKernel) // Has the random number generator been initialized?
    setupKernelCU<<<dimGrid, dimBlock>>>(deviceStates, processLoad);
  distortCU<<<dimGrid, dimBlock>>>(deviceStates, deviceTrainingSet, processLoad);

  err = hipDeviceSynchronize();
  if(err != hipSuccess)
  {
    std::cout << "Error in synchronizing distortCU " << err << std::endl;
    exit(1);
  }
}
